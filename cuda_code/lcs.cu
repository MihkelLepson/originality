#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ float CalcLcs(const int* target,
                       const int m,
                       const int* reference,
                       const int n,
                       int* L,
                       const int tid,
                       const int start_idx,
                       const int lengthL,
                       const int max_distance) {
    int offset = start_idx+tid;
    for (int i = 0; i <= m; i++) {
        for (int j = 0; j <= n; j++) {
            if (i == 0 || j == 0)
                L[i*lengthL + j+offset] = 0;
            else if (target[i - 1] == reference[j - 1])
                L[i*lengthL + j+offset] = L[(i-1)*lengthL + j-1+offset] + 1;
            else
                L[i*lengthL + j+offset] = max(L[(i-1)*lengthL + j+offset], L[i*lengthL + j-1+offset]);
        }
    }
    int len = L[m*lengthL + n+offset];
    if (len <= 1) {
        return (float)len;
    }
    // Find the distances between members of LCS.
    // In locations1 we store the locations in the reference text and in locations2 we store the locations in the target text
    int locations1[512];
    int locations2[512];

    int row = m;
    int col = n;
    int val = len;

    while (val > 0) {
        if ((col > 1) && (L[row*lengthL + (col-1)+offset] == val)) {
            col -= 1;
        } else if ((row > 1) && (L[(row-1)*lengthL + col + offset] == val)) {
            row -= 1;
        } else {
            locations1[val-1] = col;
            locations2[val-1] = row;
            val -= 1;
        }
    }
    // The lcs without taking into account distances between members is atleast 2. (We returned if was <= 1).
    // We add the rest of the members but we modify their value based on the distance.
    float lcs = 1.0;
    for (int k = 0; k < len-1; k++) {
        // We take the maximum distance of the two.
        if (locations1[k+1] - locations1[k] >= locations2[k+1] - locations2[k]) {
            // If the two members in the lcs are consecutive then there is no penalty.
            lcs =  lcs + 1.0 - (float)(min(locations1[k+1] - locations1[k] - 1, max_distance)) / max_distance;
        } else {
            lcs = lcs + 1.0 - (float)(min(locations2[k+1] - locations2[k] - 1, max_distance)) / max_distance;
        }
    }

    return lcs;
}

// CUDA kernel
__global__ void lcsKernel(int* targets,
                          int* referneces,
                          float* lcs,
                          const int* divide_points,
                          int* L,
                          const int size_tar,
                          const int size_div,
                          const int lengthL) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size_div-1) {
        int start_idx = divide_points[tid];
        int end_idx = divide_points[tid + 1];

        int subarray_size = end_idx - start_idx;
        int* reference = &referneces[start_idx];
        lcs[tid] = CalcLcs(targets, size_tar, reference, subarray_size, L, tid, start_idx, lengthL, 20);
    }
}

extern "C" {
// Entry point function to be called from Python
void cudaLcs(int* targets,
             int* referneces,
             int* lcs,
             int* divide_points_tar,
             int* divide_points_ref,
             int size_ref,
             int size_div_tar,
             int size_div_ref) {
    
    int* d_targets;
    int* d_referneces;
    float* d_lcs;
    int* d_divide_points;
    int* d_L;

    // Allocate device memory
    hipMalloc((void**)&d_referneces, sizeof(int) * size_ref);
    hipMalloc((void**)&d_lcs, sizeof(float) * (size_div_ref-1));
    hipMalloc((void**)&d_divide_points, sizeof(int) * size_div_ref); // We need one extra element for the last index

    // Copy input data from host to device
    hipMemcpy(d_referneces, referneces, sizeof(int) * size_ref, hipMemcpyHostToDevice);
    hipMemcpy(d_divide_points, divide_points_ref, sizeof(int) * size_div_ref, hipMemcpyHostToDevice); // Copy divide_points with an extra element

    // Define the kernel parameters
    int block_size = 256;
    int grid_size = (size_div_ref + block_size - 1) / block_size;

    int size_tar;
    // We process single target text at a time.
    for(int i = 0; i < size_div_tar-1; i++) {
        size_tar = divide_points_tar[i+1]-divide_points_tar[i];
        // Allocate the memory for target text.
        hipMalloc((void**)&d_targets, sizeof(int) * size_tar);
        hipMemcpy(d_targets, &targets[divide_points_tar[i]], sizeof(int) * size_tar, hipMemcpyHostToDevice);
        // Allocate the memory for dynamic programming matrix. The matrix can have more then 2**15 cells which
        // is more than the local's memory size (depends on the GPU). Hence we use the slower global memory.
        hipMalloc((void**)&d_L, sizeof(int) * (size_tar + 1)*(size_ref+size_div_ref-1));
        hipMemset(d_L, 0, sizeof(int) * (size_tar + 1)*(size_ref+size_div_ref-1));

        // Calculate the LCS with each reference text
        lcsKernel<<<grid_size, block_size>>>(d_targets, d_referneces, d_lcs, d_divide_points, d_L, size_tar, size_div_ref, size_ref+size_div_ref-1);
        hipDeviceSynchronize();
        // Check if errors occured in kernel function.
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }
        // Get results
        hipMemcpy(&lcs[i*(size_div_ref-1)], d_lcs, sizeof(float) * (size_div_ref-1), hipMemcpyDeviceToHost);

        hipFree(d_targets);
        hipFree(d_L);
    }
    
    // Free device memory
    hipFree(d_referneces);
    hipFree(d_lcs);
    hipFree(d_divide_points);
}
}
